/*
 * Author: Miras Khamit
 * Class: CSS 535
 *
 * Usage:
 *  
 *   ncu --set full ./a.out  <ArraySize> <ThreadSize> <Mode> > ...txt
 * Example:
 *   ncu --set full ./a.out  10000 32 1 > naive.txt
 *
 * Compilation:
 *     nvcc labN.cu
 */
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdio>
#include <cstdlib>


__global__ void gemv_naive(const double* A, const double* x, double* y, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        double sum = 0.0;
        for (int j = 0; j < cols; j++) {
            sum += A[row * cols + j] * x[j];
        }
        y[row] = sum;
    }
}

__global__ void gemv_global_mem_opt(const double* A, const double* x, double* y, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        double sum = 0.0;
        // Unrolling the loop to improve efficiency when accessing global memory.
        // Accessing global memory in chunks of 4 elements per iteration.
        for (int j = 0; j < cols; j += 4) {
            sum += A[row * cols + j] * x[j];
            if (j + 1 < cols) sum += A[row * cols + j + 1] * x[j + 1];
            if (j + 2 < cols) sum += A[row * cols + j + 2] * x[j + 2];
            if (j + 3 < cols) sum += A[row * cols + j + 3] * x[j + 3];
        }
        y[row] = sum;
    }
    // Global memory optimization strategy:
    // 1. We utilize loop unrolling to improve memory access efficiency, minimizing the number of global memory accesses.
    // 2. Each thread accesses 4 elements at a time (if available) in order to maximize memory throughput.
    // 3. Make sure that threads access contiguous memory locations for better memory coalescing.
}

__global__ void gemv_shared_mem(const double* A, const double* x, double* y, int rows, int cols) {
    __shared__ double x_shared[512];  // Shared memory for the vector x
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < rows) {
        double sum = 0.0;
        // Load chunks of vector 'x' into shared memory in a coalesced manner.
        for (int j = 0; j < cols; j += blockDim.x) {
            if (threadIdx.x + j < cols) {
                x_shared[threadIdx.x] = x[threadIdx.x + j];
            }
            __syncthreads();
            
            // Perform the multiplication for the block's rows and columns using shared memory.
            for (int k = 0; k < blockDim.x && (j + k) < cols; k++) {
                sum += A[row * cols + j + k] * x_shared[k];
            }
            __syncthreads();
        }
        y[row] = sum;
    }
    // Shared memory optimization strategy:
    // 1. Vector `x` is loaded into shared memory for fast access by the threads within a block.
    // 2. This eliminates redundant global memory reads and improves performance by utilizing fast shared memory.
    // 3. Threads within the block work together to load the required portion of `x`, reducing memory latency.
    // 4. Synchronization (with __syncthreads()) ensures that all threads have loaded their data before performing computations.
}

__global__ void gemv_register_opt(const double* A, const double* x, double* y, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        double sum = 0.0f;

        // Loop unrolling to increase the number of operations per thread and improve register usage.
        int j;
        for (j = 0; j <= cols - 4; j += 4) {
            // Each thread uses registers to hold intermediate results, reducing the need for slow global memory accesses.
            sum += A[row * cols + j] * x[j];
            sum += A[row * cols + j + 1] * x[j + 1];
            sum += A[row * cols + j + 2] * x[j + 2];
            sum += A[row * cols + j + 3] * x[j + 3];
        }
        
        // Handle remaining elements if cols is not a multiple of 4
        for (; j < cols; ++j) {
            sum += A[row * cols + j] * x[j];
        }

        y[row] = sum;
    }
    // Register optimization strategy:
    // 1. Loop unrolling reduces the number of iterations, increasing the instruction-level parallelism.
    // 2. By storing intermediate results in registers, we avoid costly global memory accesses.
    // 3. Register usage is particularly beneficial for small to medium-sized matrices as it eliminates memory bottlenecks.
    // 4. Optimizing with register usage helps to improve thread execution speed, minimizing register spilling and maximizing performance.
}

void run_kernel(void (*kernel)(const double*, const double*, double*, int, int), const char* name, int array_size, int block_size) {
    double *A, *x, *y;
    double *d_A, *d_x, *d_y;
    
    hipHostMalloc(&A, array_size * array_size * sizeof(double), hipHostMallocDefault);
    hipHostMalloc(&x, array_size * sizeof(double), hipHostMallocDefault);
    hipHostMalloc(&y, array_size * sizeof(double), hipHostMallocDefault);
    
    hipMalloc(&d_A, array_size * array_size * sizeof(double));
    hipMalloc(&d_x, array_size * sizeof(double));
    hipMalloc(&d_y, array_size * sizeof(double));
    
    for (int i = 0; i < array_size * array_size; i++) A[i] = static_cast<double>(rand()) / RAND_MAX;
    for (int i = 0; i < array_size; i++) x[i] = static_cast<double>(rand()) / RAND_MAX;
    
    hipMemcpy(d_A, A, array_size * array_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, array_size * sizeof(double), hipMemcpyHostToDevice);
    
    dim3 blockSize(block_size);
    dim3 gridSize((array_size + block_size - 1) / block_size);
    
    auto start = std::chrono::high_resolution_clock::now();
    kernel<<<gridSize, blockSize>>>(d_A, d_x, d_y, array_size, array_size);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    
    hipMemcpy(y, d_y, array_size * sizeof(double), hipMemcpyDeviceToHost);
    std::chrono::duration<double, std::milli> duration = end - start;
    
    std::cout << "Execution time of " << name << " with array_size=" << array_size << " and block_size=" << block_size << ": " << duration.count() << " ms" << std::endl;
    
    hipHostFree(A); hipHostFree(x); hipHostFree(y);
    hipFree(d_A); hipFree(d_x); hipFree(d_y);
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: " << argv[0] << " <ArraySize> <ThreadSize> <Mode>" << std::endl;
        return 1;
    }
    int array_size = std::atoi(argv[1]);
    int block_size = std::atoi(argv[2]);
    int mode = std::atoi(argv[3]);
    switch (mode) {
        case 1:
            run_kernel(gemv_naive, "Naive GEMV", array_size, block_size);
            break;
        case 2:
            run_kernel(gemv_shared_mem, "Shared Memory Optimized GEMV", array_size, block_size);
            break;
        case 3:
            run_kernel(gemv_global_mem_opt, "Global Memory Optimized GEMV", array_size, block_size);
            break;
        case 4:
            run_kernel(gemv_register_opt, "Register Optimized GEMV", array_size, block_size);
            break;
        default:
            std::cerr << "Invalid Mode!" << std::endl;
            return 1;
    }
    return 0;
}